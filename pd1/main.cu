#include "hip/hip_runtime.h"
#include<stdio.h>
#include<sys/time.h>

#define GRID_SIZE 8192
#define BLOCK_SIZE 256

__global__ void vectorAdd(int n, int* a, int* b, int* c)
{
	for (int i = blockIdx.x * BLOCK_SIZE + threadIdx.x; i < n;
		i += GRID_SIZE * BLOCK_SIZE) {

		c[i] = a[i] + b[i];
	}
}

int main() {
	int n, t;
	scanf("%d %d", &n, &t);

	int* a;
	int* devA;
	CUDA_CHECK_RETURN(hipHostMalloc((void**) &a, n * sizeof(int)));
	CUDA_CHECK_RETURN(hipMalloc((void**) &devA, n * sizeof(int)));

	int* b;
	int* devB;
	CUDA_CHECK_RETURN(hipHostMalloc((void**) &b, n * sizeof(int)));
	CUDA_CHECK_RETURN(hipMalloc((void**) &devB, n * sizeof(int)));

	int* c;
	int* devC;
	CUDA_CHECK_RETURN(hipHostMalloc((void**) &c, n * sizeof(int)));
	CUDA_CHECK_RETURN(hipMalloc((void**) &devC, n * sizeof(int)));

	for (int i = 0; i < n; i++) {
		a[i] = i;
		b[i] = n - i;
	}

	Timer t;
	t.startTimer();

	CUDA_CHECK_RETURN(hipMemcpy(devA, a, sizeof(int) * n, hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(devB, b, sizeof(int) * n, hipMemcpyHostToDevice));

	vectorAdd<<<GRID_SIZE, BLOCK_SIZE>>>(n, devA, devB, devC);
	CUDA_CHECK_RETURN(hipGetLastError());

	CUDA_CHECK_RETURN(hipMemcpy(c, devC, sizeof(int) * n, hipMemcpyDeviceToHost));

	printf("time: %.4f\n", t.stopTimer());

	for (int i = 0; i < n; i++) {
		if (c[i] != n) {
			printf("ERROR: c[%d] = %d\n", i, c[i]);
		}
	}

	CUDA_CHECK_RETURN(hipHostFree((void*) a));
	CUDA_CHECK_RETURN(hipFree((void*) devA));
	CUDA_CHECK_RETURN(hipHostFree((void*) b));
	CUDA_CHECK_RETURN(hipFree((void*) devB));
	CUDA_CHECK_RETURN(hipHostFree((void*) c));
	CUDA_CHECK_RETURN(hipFree((void*) devC));

	return 0;
}

