#include "hip/hip_runtime.h"
#include "util.h"

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#define H(a) (-a * log2f(a))
#define H2(a1, a2, p) (H(((float)(a1) + (p)) / ((float)(a1 + a2) + 1.0f)) + \
                       H(((float)(a2) + (1.0f - p)) / ((float)(a1 + a2) + 1.0f)))

/* Makra do sumowania tablicy 2 x 3 x 3 */
#define SUM_N3(a, n1, n2) (a[n1][n2][0] + a[n1][n2][1] + a[n1][n2][2])
#define SUM_N2(a, n1, n3) (a[n1][0][n3] + a[n1][1][n3] + a[n1][2][n3])
#define SUM_N1(a, n2, n3) (a[0][n2][n3] + a[1][n2][n3])

#define SUM_N2_N3(a, n1) (SUM_N3(a, n1, 0) + SUM_N3(a, n1, 1) + SUM_N3(a, n1, 2))
#define SUM_N1_N3(a, n2) (SUM_N3(a, 0, n2) + SUM_N3(a, 1, n2))
#define SUM_N1_N2(a, n3) (SUM_N2(a, 0, n3) + SUM_N2(a, 1, n3))

__device__ __forceinline__
unsigned int bfe(unsigned int x, unsigned int bit, unsigned int numBits) {
    unsigned int ret;
    asm("bfe.u32 %0, %1, %2, %3;" :
            "=r"(ret) : "r"(x), "r"(bit), "r"(numBits));
    return ret;
}

__device__ float compute_gig_1_2(int v1_p, int v2_p, int *vars, int *ds, int num_vars, int num_objects, float p)
{
    unsigned int count[2][3][3] = { 0 };

    for (unsigned int i = 0; i < num_objects; ++i) {
        unsigned int d = bfe(ds[i / 32], i % 32, 1);
        unsigned int v1 = bfe(vars[i * vars_width + v1_p / 16], (v1_p % 16) * 2, 2);
        unsigned int v2 = bfe(vars[i * vars_width + v2_p / 16], (v2_p % 16) * 2, 2);
        count[d][v1][v2]++;
    }

    float ig1, ig2, ig12, h_p;
    h_p = H2(SUM_N2_N3(count, 0), SUM_N2_N3(count, 1), p);
    ig1 = h_p - SUM_N1_N3(count, 0) * H2(SUM_N3(count, 0, 0), SUM_N3(count, 1, 0), p) -
                SUM_N1_N3(count, 1) * H2(SUM_N3(count, 0, 1), SUM_N3(count, 1, 1), p) -
                SUM_N1_N3(count, 2) * H2(SUM_N3(count, 0, 2), SUM_N3(count, 1, 2), p);
    ig2 = h_p - SUM_N1_N2(count, 0) * H2(SUM_N2(count, 0, 0), SUM_N2(count, 1, 0), p) -
                SUM_N1_N2(count, 1) * H2(SUM_N2(count, 0, 1), SUM_N2(count, 1, 1), p) -
                SUM_N1_N2(count, 2) * H2(SUM_N2(count, 0, 2), SUM_N2(count, 1, 2), p);
    ig12 = h_p - SUM_N1(count, 0, 0) * H2(count[0][0][0], count[1][0][0], p) -
                 SUM_N1(count, 1, 0) * H2(count[0][1][0], count[1][1][0], p) -
                 SUM_N1(count, 2, 0) * H2(count[0][2][0], count[1][2][0], p) -
                 SUM_N1(count, 0, 1) * H2(count[0][0][1], count[1][0][1], p) -
                 SUM_N1(count, 1, 1) * H2(count[0][1][1], count[1][1][1], p) -
                 SUM_N1(count, 2, 1) * H2(count[0][2][1], count[1][2][1], p) -
                 SUM_N1(count, 0, 2) * H2(count[0][0][2], count[1][0][2], p) -
                 SUM_N1(count, 1, 2) * H2(count[0][1][2], count[1][1][2], p) -
                 SUM_N1(count, 2, 2) * H2(count[0][2][2], count[1][2][2], p);

    return ig12 - ((ig1 > ig2) ? ig1 : ig2);
}

__global__ void compute_gig_kernel(int *vars, int *ds, int num_objects, int num_vars, float *r_gig, float p)
{
    /* Pomijamy całe bloki pod przekątną */
    if (blockIdx.x * blockDim.x >= (blockIdx.y + 1) * blockDim.y - 1) return;

    int v1_p = blockIdx.x * blockDim.x + threadIdx.x;
    int v2_p = blockIdx.y * blockDim.y + threadIdx.y;

    if (v1_p >= v2_p) return;
    if (v1_p >= num_vars) return;
    if (v2_p >= num_vars) return;

    const int num_v_padded = padToMultipleOf(num_vars, 16) / 4;

    r_gig[v1_p * num_vars + v2_p] = compute_gig_1_2(v1_p, v2_p, vars, ds, num_v_padded, num_objects, p);
}

struct GigStruct {
    float gig;
    int v1, v2;
};

#define BLOCK_SIZE 32

__global__ void compute_gig_wt_kernel(int *vars, int *ds, int num_objects, int num_vars,
                                      struct GigStruct *r_gig, int max_num_gig_structs, int* num_gig_structs,
                                      float p, float threshold)
{
    /* Pomijamy całe bloki pod przekątną */
    if (blockIdx.x * blockDim.x >= (blockIdx.y + 1) * blockDim.y - 1) return;

    int v1_p = blockIdx.x * blockDim.x + threadIdx.x;
    int v2_p = blockIdx.y * blockDim.y + threadIdx.y;

    __shared__ int shared_ds[1024];
    const int ds_size = padToMultipleOf(num_objects, 32);
    int load_n = threadIdx.x + blockDim.x * threadIdx.y;
    for (int i = load_n; i < ds_size; i += blockDim.x * blockDim.y)
        shared_ds[i] = ds[i];
    __syncthreads();

    if (v1_p >= v2_p) return;
    if (v1_p >= num_vars) return;
    if (v2_p >= num_vars) return;

    const int num_v_padded = padToMultipleOf(num_vars, 16) / 4;
    float gig = compute_gig_1_2(v1_p, v2_p, vars, shared_ds, num_v_padded, num_objects, p);
    if (gig < threshold) return;

    /* atomicInc() wraca do 0 po przepełnieniu */
    int num = atomicAdd(num_gig_structs, 1);
    if (num < max_num_gig_structs) {
        r_gig[num].gig = gig;
        r_gig[num].v1 = v1_p;
        r_gig[num].v2 = v2_p;
    }
}

/* Komparatory do sortowania _malejąco_ */
int compare_gig(const void *a, const void *b)
{
    if (((struct GigStruct*)a)->gig > ((struct GigStruct*)b)->gig) return -1;
    else if (((struct GigStruct*)a)->gig == ((struct GigStruct*)b)->gig) return 0;
    else return 1;
}

int compare_float(const void *a, const void *b)
{
    if (*((float*)a) > *((float*)b)) return -1;
    else if (*((float*)a) == *((float*)b)) return 0;
    else return 1;
}

int main()
{
    int num_objects, num_vars, result_size, real_result_size;
    float a_priori, threshold;

    float input, copy, random_trial_kernel, random_trial_copy, random_trial_process, main_kernel, main_copy, main_process, all;
    Timer timer;
    timer.start();

    CUDA_CALL(hipDeviceSetCacheConfig(hipFuncCachePreferL1));

    scanf("%d %d %d %f", &num_objects, &num_vars, &result_size, &a_priori);

    int *p_vars1 = (int*)malloc(sizeof(int) * num_vars);
    int *p_vars2 = (int*)malloc(sizeof(int) * num_vars);
    srand(time(NULL));
    for (int i = 0; i < num_vars; ++i) p_vars1[i] = i;
    for (int i = 0; i < 10000; ++i) {
        int a = rand();
        int b = rand();
        int tmp = p_vars1[a];
        p_vars1[a] = p_vars1[b];
        p_vars1[b] = tmp;
    }
    for (int i = 0; i < num_vars; ++i) p_vars2[p_vars1[i]] = i;

    Sync2BitArray2D vars(num_objects, padToMultipleOf(num_vars, 16));
    SyncBitArray ds(num_objects);

    /* Czytamy dane */
    {
        for (int i = 0; i < num_objects; ++i) {
            int a; scanf("%d", &a); a &= 1;
            ds.setHost(i, a);
            for (int j = 0; j < num_vars; ++j) {
                int b; scanf("%d", &b); b &= 3;
                vars.setHost(i, p_vars1[j], b);
            }
        }

        input = timer.lap();
    }

    /* Kopiujemy dane na kartę */
    {
        vars.syncToDevice();
        ds.syncToDevice();

        copy = timer.lap();
    }

    /* Wykonujemy zrandomizowaną próbę na pierwszym 10% zmiennych */
    {
        int random_trial_size = num_vars / 10;
        /* Ograniczam wielkość próby */
        if (random_trial_size > 8192)
            random_trial_size = 8192;
        float percent = (float)random_trial_size / (float)num_vars ;
        SyncArray2D<float> gig(random_trial_size, random_trial_size);

        dim3 block_size(BLOCK_SIZE, BLOCK_SIZE);
        dim3 grid_size(padToMultipleOf(random_trial_size, block_size.x) / block_size.x,
                       padToMultipleOf(random_trial_size, block_size.y) / block_size.y);
        compute_gig_kernel<<<grid_size, block_size>>>((char*)vars.getDevice(), (char*)ds.getDevice(),
                                                     num_objects, random_trial_size, (float*)gig.getDevice(), a_priori);
        CUDA_CALL(hipGetLastError());
        hipDeviceSynchronize();
        random_trial_kernel = timer.lap();

        gig.syncToHost();
        random_trial_copy = timer.lap();

        /* Przepisujemy obliczone GIG do spójnego kawałka pamięci,
           sortujemy i wybieramy odpowiedni element jako threshold */
        {
            int num_gig = 0;
            float *gig_sorted = (float*)malloc(sizeof(float) * random_trial_size * random_trial_size);
            for (int v1_p = 0; v1_p < random_trial_size; ++v1_p)
                for (int v2_p = v1_p + 1; v2_p < random_trial_size; ++v2_p)
                    gig_sorted[num_gig++] = gig.getHostEl(v1_p, v2_p);
            qsort(gig_sorted, num_gig, sizeof(float), compare_float);
            /* gig_sorted jest posortowany malejąco */
            threshold = gig_sorted[(int)((float)result_size * percent * percent)];
            free(gig_sorted);
        }

        random_trial_process = timer.lap();
    }

    /* Wykonujemy docelowe obliczenia na wszystkich zmiennych kernelem,
       który zapisuje tylko wartości większe niż threshold */
    {
        const int max_num_structs = result_size * 2;
        SyncArray<struct GigStruct> gig_structs(max_num_structs);
        SyncVar<int> num_structs;

        dim3 block_size(BLOCK_SIZE, BLOCK_SIZE);
        dim3 grid_size(padToMultipleOf(num_vars, block_size.x) / block_size.x,
                       padToMultipleOf(num_vars, block_size.y) / block_size.y);
        compute_gig_wt_kernel<<<grid_size, block_size>>>((char*)vars.getDevice(), (char*)ds.getDevice(),
                                num_objects, num_vars, (struct GigStruct*)gig_structs.getDevice(),
                                max_num_structs, num_structs.getDevice(), a_priori, threshold);
        CUDA_CALL(hipGetLastError());
        hipDeviceSynchronize();
        main_kernel = timer.lap();

        num_structs.syncToHost();
        gig_structs.syncToHost();
        main_copy = timer.lap();

        real_result_size = *num_structs.getHost();

        qsort(gig_structs.getHost(), *num_structs.getHost(), sizeof(struct GigStruct), compare_gig);

        for (int i = *num_structs.getHost() - 1; i >= 0; --i)
            printf("%f %d %d\n", gig_structs.getHostEl(i).gig, p_vars2[gig_structs.getHostEl(i).v1], p_vars2[gig_structs.getHostEl(i).v2]);

        main_process = timer.lap();
    }

    all = input + copy + random_trial_kernel + random_trial_copy + random_trial_process + main_kernel + main_copy + main_process;
    fprintf(stderr, "data: variables, objects, result_size, true result size, threshold\n");
    fprintf(stderr, "%d, %d, %d, %d, %f\n", num_vars, num_objects, result_size, real_result_size, threshold);
    fprintf(stderr, "times: input, copy, random_trial_kernel, random_trial_copy, random_trial_process, main_kernel, main_copy, main_process, all\n");
    fprintf(stderr, "%.1f, %.1f, %.1f, %.1f, %.1f, %.1f, %.1f, %.1f, %.1f\n", input, copy, random_trial_kernel,
                                    random_trial_copy, random_trial_process, main_kernel, main_copy, main_process, all);
    fprintf(stderr, "%.1f, %.1f, %.1f, %.1f, %.1f, %.1f, %.1f, %.1f\n", input / all * 100.0f, copy / all * 100.0f,
              random_trial_kernel / all * 100.0f, random_trial_copy / all * 100.0f, random_trial_process / all * 100.0f,
              main_kernel / all * 100.0f, main_copy / all * 100.0f, main_process / all * 100.0f);

    free(p_vars1);
    free(p_vars2);
    return 0;
}
