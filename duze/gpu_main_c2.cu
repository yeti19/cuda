#include "hip/hip_runtime.h"
#include "util.h"

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define H(a) (-a * log2f(a))
#define H2(a1, a2, p) (H(((float)(a1) + (p)) / ((float)(a1 + a2) + 1.0f)) + \
                       H(((float)(a2) + (1.0f - p)) / ((float)(a1 + a2) + 1.0f)))

/* Makra do sumowania tablicy 2 x 3 x 3 */
#define SUM_N3(a, n1, n2) (a[n1][n2][0] + a[n1][n2][1] + a[n1][n2][2])
#define SUM_N2(a, n1, n3) (a[n1][0][n3] + a[n1][1][n3] + a[n1][2][n3])
#define SUM_N1(a, n2, n3) (a[0][n2][n3] + a[1][n2][n3])

#define SUM_N2_N3(a, n1) (SUM_N3(a, n1, 0) + SUM_N3(a, n1, 1) + SUM_N3(a, n1, 2))
#define SUM_N1_N3(a, n2) (SUM_N3(a, 0, n2) + SUM_N3(a, 1, n2))
#define SUM_N1_N2(a, n3) (SUM_N2(a, 0, n3) + SUM_N2(a, 1, n3))

/* Format danych:
 *  - wektor wartości pierwszej zmiennej opisowej *v1s, 1 zmienna, wszystkie obiekty
 *  - wektor wartości drugiej zmiennej opisowej *v2s, 1 zmienna, wszystkie obiekty
 *  - wektor wartości zmiennych decyzyjnych *ds
 *  - ilość obiektów num_objects
 */
__device__ float compute_gig_1_2(char *v1s, char *v2s, char *ds, int num_objects, float p)
{
    int count[2][3][3] = { 0 };

    for (int i = 0; i < num_objects; ++i) {
        char d = (ds[i / 8] >> (i % 8)) & 1;
        char v1 = (v1s[i / 4] >> ((i % 4) * 2)) & 3;
        char v2 = (v2s[i / 4] >> ((i % 4) * 2)) & 3;
        count[d][v1][v2]++;
    }

    float ig1, ig2, ig12, h_p;
    h_p = H2(SUM_N2_N3(count, 0), SUM_N2_N3(count, 1), p);
    ig1 = h_p - SUM_N1_N3(count, 0) * H2(SUM_N3(count, 0, 0), SUM_N3(count, 1, 0), p) -
                SUM_N1_N3(count, 1) * H2(SUM_N3(count, 0, 1), SUM_N3(count, 1, 1), p) -
                SUM_N1_N3(count, 2) * H2(SUM_N3(count, 0, 2), SUM_N3(count, 1, 2), p);
    ig2 = h_p - SUM_N1_N2(count, 0) * H2(SUM_N2(count, 0, 0), SUM_N2(count, 1, 0), p) -
                SUM_N1_N2(count, 1) * H2(SUM_N2(count, 0, 1), SUM_N2(count, 1, 1), p) -
                SUM_N1_N2(count, 2) * H2(SUM_N2(count, 0, 2), SUM_N2(count, 1, 2), p);
    ig12 = h_p - SUM_N1(count, 0, 0) * H2(count[0][0][0], count[1][0][0], p) -
                 SUM_N1(count, 1, 0) * H2(count[0][1][0], count[1][1][0], p) -
                 SUM_N1(count, 2, 0) * H2(count[0][2][0], count[1][2][0], p) -
                 SUM_N1(count, 0, 1) * H2(count[0][0][1], count[1][0][1], p) -
                 SUM_N1(count, 1, 1) * H2(count[0][1][1], count[1][1][1], p) -
                 SUM_N1(count, 2, 1) * H2(count[0][2][1], count[1][2][1], p) -
                 SUM_N1(count, 0, 2) * H2(count[0][0][2], count[1][0][2], p) -
                 SUM_N1(count, 1, 2) * H2(count[0][1][2], count[1][1][2], p) -
                 SUM_N1(count, 2, 2) * H2(count[0][2][2], count[1][2][2], p);

    //printf("  IG(v1) = %f\n", ig1);
    //printf("  IG(v2) = %f\n", ig2);
    //printf("  IG(v1 u v2) = %f\n", ig12);

    return ig12 - ((ig1 > ig2) ? ig1 : ig2);
}

/* Format danych:
 *  - macierz wartości zmiennych opisowych *vars, 1 wiersz - 1 zmienna
 *  - wektor wartości zmiennych decyzyjnych *ds
 *  - ilość obiektów num_objects
 *  - ilość zmiennych num_vars
 *  - wynikowe GIG
 */
__global__ void compute_gig_kernel(char *vars, char *ds, int num_objects, int num_vars, float *r_gig, float p)
{
    int v1_p = blockIdx.x * blockDim.x + threadIdx.x;
    int v2_p = blockIdx.y * blockDim.y + threadIdx.y;

    if (v1_p >= v2_p) return;
    if (v1_p >= num_vars) return;
    if (v2_p >= num_vars) return;
    //printf("compute_gig(%d, %d) %d\n", v1_p, v2_p, blockIdx.y);
    const int num_o_padded = (num_objects - 1) / 4 + 1;

    r_gig[v1_p * num_vars + v2_p] = compute_gig_1_2(&vars[v1_p * num_o_padded], &vars[v2_p * num_o_padded], ds, num_objects, p);
    //printf(" GIG = %f\n", r_gig[v1_p * num_vars + v2_p]);
}

struct GigStruct {
    float gig;
    int v1, v2;
};

__global__ void compute_gig_wt_kernel(char *vars, char *ds, int num_objects, int num_vars,
                                      struct GigStruct *r_gig, int max_num_gig_structs, int* num_gig_structs,
                                      float p, float threshold)
{
    int v1_p = blockIdx.x * blockDim.x + threadIdx.x;
    int v2_p = blockIdx.y * blockDim.y + threadIdx.y;

    if (v1_p >= v2_p) return;
    if (v1_p >= num_vars) return;
    if (v2_p >= num_vars) return;
    //printf("compute_gig(%d, %d) %d\n", v1_p, v2_p, blockIdx.y);

    const int num_o_padded = (num_objects - 1) / 4 + 1;
    float gig = compute_gig_1_2(&vars[v1_p * num_o_padded], &vars[v2_p * num_o_padded], ds, num_objects, p);
    if (gig < threshold) return;
    /* atomicInc() wraps around to 0 */
    int num = atomicAdd(num_gig_structs, 1);
    if (num < max_num_gig_structs) {
        r_gig[num].gig = gig;
        r_gig[num].v1 = v1_p;
        r_gig[num].v2 = v2_p;
    }
    //printf(" GIG = %f\n", r_gig[v1_p * num_vars + v2_p]);
}

/* Komparatory do sortowania _malejąco_ */
int compare_gig(const void *a, const void *b)
{
    if (((struct GigStruct*)a)->gig > ((struct GigStruct*)b)->gig) return -1;
    else if (((struct GigStruct*)a)->gig == ((struct GigStruct*)b)->gig) return 0;
    else return 1;
}

int compare_float(const void *a, const void *b)
{
    if (*((float*)a) > *((float*)b)) return -1;
    else if (*((float*)a) == *((float*)b)) return 0;
    else return 1;
}

int main()
{
    int num_objects, num_vars, result_size, real_result_size;
    float a_priori, threshold;

    float input, copy, random_trial_kernel, random_trial_copy, random_trial_process, main_kernel, main_copy, main_process, all;
    Timer timer;
    timer.start();

    scanf("%d %d %d %f", &num_objects, &num_vars, &result_size, &a_priori);

    Sync2BitArray2D vars(num_vars, num_objects);
    SyncBitArray ds(num_objects);

    /* Czytamy dane */
    {
        for (int i = 0; i < num_objects; ++i) {
            int a; scanf("%d", &a); a &= 1;
            ds.setHost(i, a);
            for (int j = 0; j < num_vars; ++j) {
                int b; scanf("%d", &b); b &= 3;
                vars.setHost(j, i, b);
            }
        }

        input = timer.lap();
    }

    /* Kopiujemy dane na kartę */
    {
        vars.syncToDevice();
        ds.syncToDevice();

        copy = timer.lap();
    }

    /* Wykonujemy zrandomizowaną próbę na pierwszym 10% zmiennych */
    {
        int random_trial_size = num_vars / 10;
        /* Alokacja pamięci na wynikowe GIG się nie udaje gdy pamięć jest > ok. 400MB.
           XXX: Tablica gig nie musiałaby być kwadratowa. */
        if (random_trial_size > 8192)
            random_trial_size = 8192;
        float percent = (float)random_trial_size / (float)num_vars ;
        SyncArray2D<float> gig(random_trial_size, random_trial_size);

        dim3 block_size(16, 16);
        dim3 grid_size(padToMultipleOf(random_trial_size, block_size.x) / block_size.x,
                       padToMultipleOf(random_trial_size, block_size.y) / block_size.y);
        compute_gig_kernel<<<grid_size, block_size>>>((char*)vars.getDevice(), (char*)ds.getDevice(),
                                                     num_objects, random_trial_size, (float*)gig.getDevice(), a_priori);
        CUDA_CALL(hipGetLastError());
        hipDeviceSynchronize();
        random_trial_kernel = timer.lap();

        gig.syncToHost();
        random_trial_copy = timer.lap();

        /* Przepisujemy obliczone GIG do spójnego kawałka pamięci,
           sortujemy i wybieramy odpowiedni element jako threshold */
        {
            int num_gig = 0;
            float *gig_sorted = (float*)malloc(sizeof(float) * num_vars * num_vars);
            for (int v1_p = 0; v1_p < random_trial_size; ++v1_p)
                for (int v2_p = v1_p + 1; v2_p < random_trial_size; ++v2_p)
                    gig_sorted[num_gig++] = gig.getHostEl(v1_p, v2_p);
            qsort(gig_sorted, num_gig, sizeof(float), compare_float);
            /* gig_sorted jest posortowany malejąco */
            threshold = gig_sorted[(int)((float)result_size * percent * percent)];
            free(gig_sorted);
        }

        random_trial_process = timer.lap();
    }

    /* Wykonujemy docelowe obliczenia na wszystkich zmiennych kernelem,
       który zapisuje tylko wartości większe niż threshold */
    {
        const int max_num_structs = result_size * 2;
        SyncArray<struct GigStruct> gig_structs(max_num_structs);
        SyncVar<int> num_structs;

        dim3 block_size(16, 16);
        dim3 grid_size(padToMultipleOf(num_vars, block_size.x) / block_size.x,
                       padToMultipleOf(num_vars, block_size.y) / block_size.y);
        compute_gig_wt_kernel<<<grid_size, block_size>>>((char*)vars.getDevice(), (char*)ds.getDevice(),
                                num_objects, num_vars, (struct GigStruct*)gig_structs.getDevice(),
                                max_num_structs, num_structs.getDevice(), a_priori, threshold);
        CUDA_CALL(hipGetLastError());
        hipDeviceSynchronize();
        main_kernel = timer.lap();

        num_structs.syncToHost();
        gig_structs.syncToHost();
        main_copy = timer.lap();

        real_result_size = *num_structs.getHost();

        qsort(gig_structs.getHost(), *num_structs.getHost(), sizeof(struct GigStruct), compare_gig);

        for (int i = *num_structs.getHost() - 1; i >= 0; --i)
            printf("%f %d %d\n", gig_structs.getHostEl(i).gig, gig_structs.getHostEl(i).v1, gig_structs.getHostEl(i).v2);

        main_process = timer.lap();
    }

    all = input + copy + random_trial_kernel + random_trial_copy + random_trial_process + main_kernel + main_copy + main_process;
    fprintf(stderr, "data: variables, objects, result_size, true result size, threshold\n");
    fprintf(stderr, "%d, %d, %d, %d, %f\n", num_vars, num_objects, result_size, real_result_size, threshold);
    fprintf(stderr, "times: input, copy, random_trial_kernel, random_trial_copy, random_trial_process, main_kernel, main_copy, main_process, all\n");
    fprintf(stderr, "%.1f, %.1f, %.1f, %.1f, %.1f, %.1f, %.1f, %.1f, %.1f\n", input, copy, random_trial_kernel,
                                    random_trial_copy, random_trial_process, main_kernel, main_copy, main_process, all);
    fprintf(stderr, "%.1f, %.1f, %.1f, %.1f, %.1f, %.1f, %.1f, %.1f\n", input / all * 100.0f, copy / all * 100.0f,
              random_trial_kernel / all * 100.0f, random_trial_copy / all * 100.0f, random_trial_process / all * 100.0f,
              main_kernel / all * 100.0f, main_copy / all * 100.0f, main_process / all * 100.0f);

    return 0;
}
